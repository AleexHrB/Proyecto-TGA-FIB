#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <sys/times.h>
#include <sys/resource.h>

void CheckCudaError(char sms[], int line) {
  hipError_t error;
 
  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }

  else {
	  printf("Ok jefe");
  }


}


__global__ void insertion(int *v, unsigned int n, unsigned int size_i) {
	unsigned int id = threadIdx.x + blockDim.x * blockIdx.x;

	unsigned int lim_inf = id * size_i;

	unsigned int i = lim_inf;
	unsigned int lim_sup = lim_inf + size_i;

	//Insertion Sort a bloques de size_i de tamaño
	while (i < lim_sup) {
		int x = v[i];
		int j = i - 1;

		while (j >= (int)lim_inf && v[j] > x) {
			v[j+1] = v[j];
			--j;
		}
		v[j+1] = x;
		++i;
	}

	
}
__global__ void PathMerge(int *v, unsigned int n, int *res, unsigned int size_i, unsigned int i) {

		//Threads que usas en cada nivel del arbol por merge
		unsigned int num_threads = i/2;
		unsigned int id = threadIdx.x + blockDim.x * blockIdx.x;
		//Asignar el id a la parte del vector correspondiente
		unsigned int part = (id - id%num_threads) / num_threads;
		//Dentro de la seccion del vector, identificar los threads 
		unsigned int id_ins = id%num_threads;

		//Posicion Inicial del Vector
		unsigned int beg = part*size_i*i;
		//Posicion Final del Vector
		unsigned int end = beg + size_i * i  - 1;
		//Posicion del Medio
		unsigned int mid = beg + size_i * (i/2) - 1;
		//Los vectores ordenados van de v[beg ... mid], v[mid+1...end]

		//Tamaño de un vector
		unsigned int n_ind = mid - beg + 1;
		//La Diagonal que le corresponde
		unsigned int DiagNum = id_ins * 2 * n_ind /num_threads;

		unsigned int st[2], ed[2], pt[2];
		//Punto inicio de la diagonal
		st[0] = DiagNum > n_ind ? n_ind : DiagNum;
		st[1] = DiagNum > n_ind ? DiagNum - n_ind : 0;
		st[0] += beg;
		st[1] += beg;

		//Punto final de la diagonal
		ed[0] = st[1];
		ed[1] = st[0];

		//Punto medio para hacer busqueda binaria
		pt[1] = (st[1] + ed[1]) / 2;
		pt[0] = st[0] - (pt[1] - st[1]);
		//Busqueda binaria mientras haya dos casillas en la diagonal
		while (st[1] + 1 < ed[1]) {
			pt[1] = (st[1] + ed[1]) / 2;
			pt[0] = st[0] - (pt[1] - st[1]);

			//Sumar n_ind para coger los elementos del otro vector
			if (v[pt[0]] > v[pt[1] + n_ind - 1]) {
				if (v[pt[0] - 1] <= v[pt[1] + n_ind]) break;
				else {
					st[0] = pt[0];
					st[1] = pt[1];
				}
			}

			else {
				ed[0] = pt[0];
				ed[1] = pt[1];
			}
		}

		//En caso de que haya una solo casilla, decidir (si has llegado hasta aqui deberias de solo mirar los extremos de las diagonales)
		if (ed[1] - st[1] == 1) {
			if (v[st[0] - 1] <=  v[st[1] + n_ind]) {
				pt[0] = st[0];
				pt[1] = st[1];
			}

			else {
				pt[0] = ed[0];
				pt[1] = ed[1];
			}
		}


		//Cada thread hace un numero igual de iteraciones
		unsigned int aux = 0;
		unsigned int steps = 2 * n_ind/num_threads;

		//Posicion del vector para comenzar
		unsigned int idx = beg + DiagNum;
		unsigned int j = pt[0];
		unsigned int k = pt[1] + n_ind;



		//Merge normal
		while (aux < steps && j <= mid && k <= end) {

			if (v[j] <= v[k]) {
				res[idx] = v[j];
				++idx;
				++j;
			}

			else {
				res[idx] = v[k];
				++idx;
				++k;
			}

			++aux;
		}
		
		while(aux < steps && j <= mid) {
			res[idx] = v[j];
			++idx;
			++j;
			++aux;
		}

		while(aux < steps && k <= end) {
			res[idx] = v[k];
			++idx;
			++k;
			++aux;
		}

}




int main(int argc, char** argv) {

	if (argc != 5) {
		printf("Número de parámetros no válido\n");
		return -1;
	}
	
	int *d_v;
	unsigned int n = 1 << atoi(argv[1]);
	unsigned int size_i = 1 << atoi(argv[2]);
	int *h_v = (int *)malloc(n*sizeof(int));
	int *d_aux;
	srand(21364);
	hipEvent_t e1, e2, e3, e4, e5, e6;
	hipEventCreate(&e1);
	hipEventCreate(&e2);
	hipEventCreate(&e3);
	hipEventCreate(&e4);
	hipEventCreate(&e5);
	hipEventCreate(&e6);

	unsigned int threads = n / size_i;
	const unsigned int block_size = 1024 / (1 << atoi(argv[4]));
	unsigned int block = threads / block_size;

	if (block == 0) block = 1;
	

	const unsigned int sorted_mode = 0;
	const unsigned int random_mode = 1;
	const unsigned int sorted_back_mode = 2;
	const unsigned int mode = atoi(argv[3]);


	if (mode == random_mode) for (unsigned int i = 0; i < n; ++i) h_v[i] = rand();
	else if (mode == sorted_mode) for (unsigned int i = 0; i < n; ++i) h_v[i] = i;
	else for (unsigned int i = 0; i < n; ++i) h_v[i] = n - i;

	hipMalloc((void **)&d_v, n*sizeof(int));
	hipMalloc((void **)&d_aux, n*sizeof(int));

	hipEventRecord(e1, 0);
	hipMemcpyAsync(d_v, h_v, n*sizeof(int), hipMemcpyHostToDevice);
	hipEventRecord(e2, 0);
	hipEventSynchronize(e2);
	float HtD_t;
	hipEventElapsedTime(&HtD_t, e1, e2);

	hipEventRecord(e3, 0);
	insertion<<<block, threads/block>>>(d_v,n, size_i);
	unsigned int b = 0;

	for (unsigned int i = 2; i <= n/size_i; i *= 2) {
		unsigned int threads_merge = threads / i;
		unsigned int block_merge = threads_merge / block_size;
		if (block_merge == 0) block_merge = 1;
		if (!b) PathMerge<<<block_merge, threads_merge/block_merge>>>(d_v, n, d_aux, size_i, i);
		else PathMerge<<<block_merge, threads_merge/block_merge>>>(d_aux, n, d_v, size_i, i);
		b = !b;
	}
	hipEventRecord(e4, 0);
	hipEventSynchronize(e4);
	float kernel_t;
	hipEventElapsedTime(&kernel_t, e3, e4);

	hipEventRecord(e5, 0);
	if (b) hipMemcpyAsync(h_v, d_aux, n*sizeof(int), hipMemcpyDeviceToHost);
	else hipMemcpyAsync(h_v, d_v, n*sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(e6, 0);
	hipEventSynchronize(e6);
	float DtH_t;
	hipEventElapsedTime(&DtH_t, e5, e6);
	//for (unsigned int i = 0; i < n; ++i) printf("%d\n", h_v[i]);

	char s[10];
	if (mode == random_mode) strcpy(s, "Random");
	else if (mode == sorted_mode) strcpy(s, "Ordenado");
	else strcpy(s, "Al Revés");

	printf("Modo: %s\n", s); 
	printf("n: %d, size_i: %d\n", n, size_i); 
	printf("Tiempo Kernels: %f ms\n", kernel_t); 
	printf("Ancho de Banda HtD: %f GB/s, Ancho de Banda Kernels: %f GB/s, Ancho de Banda DtH: %f GB/s\n", (n*sizeof(unsigned int)) / (HtD_t * 1e6), (n*sizeof(unsigned int)) / ((kernel_t+DtH_t) * 1e6), (n*sizeof(unsigned int)) / (DtH_t * 1e6));

	free(h_v);
	hipFree(d_v);
	hipFree(d_aux);
}

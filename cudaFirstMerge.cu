#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <sys/times.h>
#include <sys/resource.h>


void CheckCudaError(char sms[], int line) {
  hipError_t error;
 
  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }

  else {
	  printf("Ok jefe");
  }


}


__global__ void insertion(int *v, unsigned int n, unsigned int size_i) {
	unsigned int id = threadIdx.x + blockDim.x * blockIdx.x;

	unsigned int lim_inf = id * size_i;

	unsigned int i = lim_inf;
	unsigned int lim_sup = lim_inf + size_i;

	//Insertion Sort a bloques de size_i de tamaño
	while (i < lim_sup) {
		int x = v[i];
		int j = i - 1;

		while (j >= (int)lim_inf && v[j] > x) {
			v[j+1] = v[j];
			--j;
		}
		v[j+1] = x;
		++i;
	}

	
}

__global__ void merge(int *v, unsigned int n, int* res, unsigned int size_i, unsigned int i) {

		unsigned int id = threadIdx.x + blockDim.x * blockIdx.x;
		unsigned int beg = id*size_i*i;

		unsigned int end = beg + size_i * i  - 1;
		unsigned int mid = beg + size_i * (i/2) - 1;
		unsigned int j = beg;
		unsigned int k = mid + 1;
		unsigned int idx = beg;

		while (j <= mid && k <= end) {

			if (v[j] <= v[k]) {
				res[idx] = v[j];
				++idx;
				++j;
			}

			else {
				res[idx] = v[k];
				++idx;
				++k;
			}
		}
		
		while(j <= mid) {
			res[idx] = v[j];
			++idx;
			++j;
		}

		while(k <= end) {
			res[idx] = v[k];
			++idx;
			++k;
		}
	
}

int main(int argc, char** argv) {

	if (argc != 4) {
		printf("Número de parámetros no válido\n");
		return -1;
	}
	
	int *d_v;
	unsigned int n = 1 << atoi(argv[1]);
	unsigned int size_i = 1 << atoi(argv[2]);
	int *h_v = (int *)malloc(n*sizeof(int));
	int *d_aux;
	srand(21364);
	hipEvent_t e1, e2, e3, e4, e5, e6;
	hipEventCreate(&e1);
	hipEventCreate(&e2);
	hipEventCreate(&e3);
	hipEventCreate(&e4);
	hipEventCreate(&e5);
	hipEventCreate(&e6);

	unsigned int threads = n / size_i;
	unsigned int block = threads / 1024;

	if (block == 0) block = 1;
	

	const unsigned int sorted_mode = 0;
	const unsigned int random_mode = 1;
	const unsigned int sorted_back_mode = 2;
	const unsigned int mode = atoi(argv[3]);


	if (mode == random_mode) for (unsigned int i = 0; i < n; ++i) h_v[i] = rand();
	else if (mode == sorted_mode) for (unsigned int i = 0; i < n; ++i) h_v[i] = i;
	else for (unsigned int i = 0; i < n; ++i) h_v[i] = n - i;

	hipMalloc((void **)&d_v, n*sizeof(int));
	hipMalloc((void **)&d_aux, n*sizeof(int));

	hipEventRecord(e1, 0);
	hipMemcpyAsync(d_v, h_v, n*sizeof(int), hipMemcpyHostToDevice);
	hipEventRecord(e2, 0);
	hipEventSynchronize(e2);
	float HtD_t;
	hipEventElapsedTime(&HtD_t, e1, e2);

	hipEventRecord(e3, 0);
	insertion<<<block, threads/block>>>(d_v,n, size_i);
	unsigned int b = 0;

	for (unsigned int i = 2; i <= n/size_i; i *= 2) {
		unsigned int threads_merge = threads / i;
		unsigned int block_merge = threads_merge / 1024;
		if (block_merge == 0) block_merge = 1;
		if (!b) merge<<<block_merge, threads_merge/block_merge>>>(d_v, n, d_aux, size_i, i);
		else merge<<<block_merge, threads_merge/block_merge>>>(d_aux, n, d_v, size_i, i);
		b = !b;
	}
	hipEventRecord(e4, 0);
	hipEventSynchronize(e4);
	float kernel_t;
	hipEventElapsedTime(&kernel_t, e3, e4);

	hipEventRecord(e5, 0);
	if (b) hipMemcpyAsync(h_v, d_aux, n*sizeof(int), hipMemcpyDeviceToHost);
	else hipMemcpyAsync(h_v, d_v, n*sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(e6, 0);
	hipEventSynchronize(e6);
	float DtH_t;
	hipEventElapsedTime(&DtH_t, e5, e6);
	//for (unsigned int i = 0; i < n; ++i) printf("%d\n", h_v[i]);

	char s[10];
	if (mode == random_mode) strcpy(s, "Random");
	else if (mode == sorted_mode) strcpy(s, "Ordenado");
	else strcpy(s, "Al Revés");

	printf("Modo: %s\n", s); 
	printf("n: %d, size_i: %d\n", n, size_i); 
	printf("Tiempo Kernels: %f ms\n", kernel_t); 
	printf("Ancho de Banda HtD: %f GB/s, Ancho de Banda Kernels: %f GB/s, Ancho de Banda DtH: %f GB/s\n", (n*sizeof(unsigned int)) / (HtD_t * 1e6), (n*sizeof(unsigned int)) / ((kernel_t+DtH_t) * 1e6), (n*sizeof(unsigned int)) / (DtH_t * 1e6));

	free(h_v);
	hipFree(d_v);
	hipFree(d_aux);
}
